#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "../yuv_yuv420.h"
__global__ void
color_convert_kernel(unsigned char *output, unsigned char *input, 
	int sourceWidth, int sourceHeight, int blockid_count, int threadid_count)
{
	yuyv_yuv420_color_convert(blockIdx.x, threadIdx.x, output, input, 
		sourceWidth, sourceHeight, blockid_count, threadid_count);
	return;
}

int
main(int argc, char **argv) 
{
	int count=1;
	static struct timeval cur_tv,prev_tv;

	char *input_image = argv[1];
	int in_width = atoi(argv[2]);
	int in_height = atoi(argv[3]);

	char *output_image = argv[4];
	int out_width = atoi(argv[5]);
	int out_height = atoi(argv[6]);

	char *src_buffer;
	char *des_buffer;

	char *src_buffer_cu;
	char *des_buffer_cu;

	FILE *fp;

	checkCudaErrors(hipHostAlloc((void **)&src_buffer, in_width*in_height*2, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc((void **)&des_buffer, out_width*out_height*1.5, hipHostMallocMapped));

	checkCudaErrors(hipHostGetDevicePointer((void **)&src_buffer_cu, (void *)src_buffer, 0));
	checkCudaErrors(hipHostGetDevicePointer((void **)&des_buffer_cu, (void *)des_buffer, 0));

	fp = fopen(input_image,"r+");
	fread(src_buffer,1,in_width*in_height*2,fp);
	fclose(fp);

	printf("in_width, %d  in_height, %d out_width, %d out_height %d \n", in_width, in_height, out_width, out_height);

while (count--) {
	gettimeofday(&prev_tv, NULL);
	color_convert_kernel<<<192, 192>>>( (unsigned char *)des_buffer_cu, (unsigned char *) src_buffer_cu, in_width, in_height, 192, 192);
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&cur_tv, NULL);
	printf("Time prev_tv %lu.%06lu cur_tv %lu.%06lu = %lu\n",(unsigned long int)prev_tv.tv_sec,
								(unsigned long int)prev_tv.tv_usec,
								(unsigned long int)cur_tv.tv_sec,
								(unsigned long int)cur_tv.tv_usec,
								(unsigned long int)(((cur_tv.tv_sec-prev_tv.tv_sec)*1000000)
								+cur_tv.tv_usec-prev_tv.tv_usec));
}
	fp = fopen(output_image,"w+");
	fwrite(des_buffer,1,out_width*out_height*1.5,fp);
	fclose(fp);

	checkCudaErrors(hipHostFree(src_buffer));
        checkCudaErrors(hipHostFree(des_buffer));
}

